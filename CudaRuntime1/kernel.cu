#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <random>


// CUDA kernel to calculate Black-Scholes formula and average stock price
__global__ void blackScholesKernel(float* prices, float* strikes, float* times, float* risks, float* vols, float* results, int numOptions, int numSims, float* avgPrice) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numOptions) {
        float sum = 0.0f;
        for (int j = 0; j < numSims; j++) {
            // Generate random numbers on GPU
            hiprandState rng;
            hiprand_init(i * numSims + j, 0, 0, &rng);
            float Z = sqrt(-2.0 * log(hiprand_uniform(&rng))) * cos(2.0 * 3.1416 * hiprand_uniform(&rng));
            float St = prices[i] * exp((risks[i] - 0.5 * vols[i] * vols[i]) * times[i] + vols[i] * sqrt(times[i]) * Z);
            float payoff = fmax(St - strikes[i], 0.0f);
            sum += payoff;
        }
        results[i] = exp(-risks[i] * times[i]) * sum / numSims;

        // Calculate average stock price
        atomicAdd(avgPrice, prices[i]);
    }
}

int main() {

    // Initialize input data
    int numOptions = 10000;
    int numSims = 100;
    float* prices;
    float* strikes;
    float* times;
    float* risks;
    float* vols;
    float* results;
    hipMallocManaged(&prices, numOptions * sizeof(float));
    hipMallocManaged(&strikes, numOptions * sizeof(float));
    hipMallocManaged(&times, numOptions * sizeof(float));
    hipMallocManaged(&risks, numOptions * sizeof(float));
    hipMallocManaged(&vols, numOptions * sizeof(float));
    hipMallocManaged(&results, numOptions * sizeof(float));
    float* avgPrice;
    hipMallocManaged(&avgPrice, sizeof(float));
    *avgPrice = 0.0f;

    // Initialize input data arrays with random values
    for (int i = 0; i < numOptions; i++) {
        prices[i] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
        strikes[i] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
        times[i] = 5.0f * static_cast<float>(rand()) / RAND_MAX;
        risks[i] = 0.1f * static_cast<float>(rand()) / RAND_MAX;
        vols[i] = 0.5f * static_cast<float>(rand()) / RAND_MAX;
    }

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numOptions + threadsPerBlock - 1) / threadsPerBlock;
    blackScholesKernel << <blocksPerGrid, threadsPerBlock >> > (prices, strikes, times, risks, vols, results, numOptions, numSims, avgPrice);
    hipDeviceSynchronize();

    // Calculate average stock price
    *avgPrice /= numOptions;
    std::cout << "Average stock price: " << *avgPrice << std::endl;

    // Print results
    for (int i = 0; i < numOptions; i++) {
        std::cout << "Option " << i << ": " << results[i] << std::endl;
    }

    // Clean up
    hipFree(prices);
    hipFree(strikes);
    hipFree(times);
    hipFree(risks);
    hipFree(vols);
    hipFree(results);
    hipFree(avgPrice);

    return 0;
}
